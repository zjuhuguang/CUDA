#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>
#include <stdint.h>
#include <stdarg.h>



#include <string>
#include <sys/time.h> // for clock_gettime()
#include <unistd.h> // for usleep()
//	#define N 1008559420525856281
//#define N 0x6926C73F919FA3E7LL

//#define N 0xB8C8CBD2DAEE7DLL

__global__ void pollardKernel(int64_t m, int64_t num, int64_t* result_d, float* devData);
__global__ void cpyFunction(float* devData, int64_t* xd, int64_t m);


static double t0=0;
double Elapsed(void)
{
#ifdef _WIN32
    //  Windows version of wall time
    LARGE_INTEGER tv,freq;
    QueryPerformanceCounter((LARGE_INTEGER*)&tv);
    QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
    double t = tv.QuadPart/(double)freq.QuadPart;
#else
    //  Unix/Linux/OSX version of wall time
    struct timeval tv;
    gettimeofday(&tv,NULL);
    double t = tv.tv_sec+1e-6*tv.tv_usec;
#endif
    double s = t-t0;
    t0 = t;
    return s;
}
///////////////////////////////////////////////////////////////////////////////////////////////
// device version of f(x)
__device__ __forceinline__ int64_t fx_d(int64_t x, int64_t a, int64_t c) {
    return ( a * x * x + c);
}

// device version of binary gcd
__device__ int64_t gcd_d(int64_t u, int64_t v)
{
    int shift;
    
    /* GCD(0,v) == v; GCD(u,0) == u, GCD(0,0) == 0 */
    if (u == 0) return v;
    if (v == 0) return u;
    
    /* Let shift := lg K, where K is the greatest power of 2
     dividing both u and v. */
    for (shift = 0; ((u | v) & 1) == 0; ++shift) {
        u >>= 1;
        v >>= 1;
    }
    
    while ((u & 1) == 0)
        u >>= 1;
    
    /* From here on, u is always odd. */
    do {
        /* remove all factors of 2 in v -- they are not common */
        /*   note: v is not zero, so while will terminate */
        while ((v & 1) == 0)  /* Loop X */
            v >>= 1;
        
        /* Now u and v are both odd. Swap if necessary so u <= v,
         then set v = v - u (which is even). For bignums, the
         swapping is just pointer movement, and the subtraction
         can be done in-place. */
        if (u > v) {
            int64_t t = v; v = u; u = t;}  // Swap u and v.
        v = v - u;                       // Here v >= u.
    } while (v != 0);
    
    /* restore common factors of 2 */
    return u << shift;
}
//////////////////////////////////////////////////////////////////////////////////////////////




__global__ void pollardKernel(int64_t m, int64_t num, int64_t* result_d, int64_t* xd)
{	
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	int64_t z, d = 1;
	int64_t x = xd[4 * threadId];
	int64_t y = xd[4 * threadId + 1];
	int64_t a = xd[4 * threadId + 2];
	int64_t c = xd[4 * threadId + 3];
	
    x = fx_d(x,a,c) % num;
    y = fx_d(fx_d(y,a,c),a,c) % num;
    z = abs(x-y);
    d = gcd_d(z,num);
    
    // copy updated state back into global memory
    xd[threadId * 4] = x;
    xd[threadId * 4 + 1] = y;

    // test to see if it found a factor
    if (d != 1 && d != num )
    {
        // if found, copy it into global syncronization variable "found"
        *result_d = d;
    }
 //   __syncthreads();
}


__global__ void cpyFunction(float* devData, int64_t* xd, int64_t m)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	xd[4 * threadId] = 0;
	xd[4 * threadId + 1] = 0;
	xd[4 * threadId + 2] = (int64_t)(devData[threadId * 2] * m);
	xd[4 * threadId + 3] = (int64_t)(devData[threadId * 2 + 1] * m);
}




int main(void)
{
	int64_t N =7576962498937463783;

    	int64_t m = sqrt(N);
    	int64_t result = 0;
  Elapsed();
//use hiprand to generate random parrallel
  size_t n =256 * 256;
  size_t i;
  hiprandGenerator_t gen;
  float *devData , *hostData;
  int64_t *xd;
  
  /* Allocate n floats on host */
  hostData = (float *) calloc(2 * n, sizeof(float));
  int64_t *x;
  /* Allocate n floats on device */
  hipMalloc((void **) &devData, 2 * n * sizeof(float));
  x = (int64_t *) calloc(4 * n, sizeof(int64_t));
  hipMalloc((void **) &xd, 4 * n*sizeof(int64_t));
  /* Create a Mersenne Twister pseudorandom number generator */
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);

  /* Set seed */
  hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
  
  /* Generate n floats on device */
  hiprandGenerateUniform(gen, devData, 2 * n);
  
  /* Copy device memory to host */
 hipMemcpy(hostData , devData , 2 * n * sizeof(float), hipMemcpyDeviceToHost);
/*
  for(i = 0; i < n; i++) {
    printf(" %1.4f\n", hostData[i]);
  }
*/
//allocate result
	int64_t* result_d;
    	hipMalloc((void**)&result_d, sizeof(int64_t));
   	hipMemcpy(result_d,&result, sizeof(int64_t), hipMemcpyHostToDevice);
	cpyFunction<<<256, 256>>>(devData, xd, m);
	hipDeviceSynchronize();
	hipMemcpy(x, xd, 4 * n * sizeof(int64_t), hipMemcpyDeviceToHost);
	hipFree(devData);

 for(i = 0; i < 4 * n; i++) {
    printf(" %d\n", x[i]);
  }
	hipDeviceSynchronize();
//call the kernel
	do{
		pollardKernel<<<256, 256>>>(m, N, result_d, xd);
		hipDeviceSynchronize();
		hipMemcpy(&result,result_d,sizeof(int64_t), hipMemcpyDeviceToHost);
	  }
	while(result == 0);
	double Td = Elapsed();
	printf("%x\n%f", result, Td);
	hipFree(result_d);
	hipFree(xd);
//	free(result);

}

















